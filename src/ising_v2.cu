#include "hip/hip_runtime.h"
#include "../inc/ising.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#define BLOCK_SIZE  16
#define GRID_SIZE 4

//functions
__global__ void calculateNewSpinKernel(int * M, int * newM, double * w, int n, int * flag);


void ising( int *G, double *w, int k , int n)
{
  struct timeval startwtime, endwtime;
  double time = 0;
  //flag to terminate if no changes are made
  int   terminate_flag;
  int * d_terminate_flag;
  //for pointer swap
  int * temp;

  //cuda
  int  * d_G, *d_newG;
  double * d_w;

  //cuda mallocs
  hipMalloc(&d_terminate_flag,sizeof(int));
  hipMalloc(&d_G, n*n*sizeof(int));
  hipMalloc(&d_newG, n*n*sizeof(int));
  hipMalloc(&d_w, 5*5*sizeof(double));

  //cuda memcpy G and W
  hipMemcpy(d_G, G, n*n*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_w, w, 5*5*sizeof(double), hipMemcpyHostToDevice);

  //declare block size and grid size
  dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 blocksPerGrid(GRID_SIZE, GRID_SIZE);



  //k steps iterations
  for(int i= 0 ; i < k ;i++)
  {
    //reset flag value
    terminate_flag = 1;
    hipMemcpy(d_terminate_flag, &terminate_flag,sizeof(int), hipMemcpyHostToDevice);
    //call kernel
    gettimeofday (&startwtime, NULL);
    calculateNewSpinKernel<<<blocksPerGrid, threadsPerBlock>>>(d_G,d_newG,d_w,n,d_terminate_flag);
    hipDeviceSynchronize();
    gettimeofday (&endwtime, NULL);
    time += (double)((endwtime.tv_usec - startwtime.tv_usec)
        /1.0e6 + endwtime.tv_sec - startwtime.tv_sec);
    //swap pointers
    temp = d_G;
    d_G = d_newG;
    d_newG = temp;  

    //we need device value for flag here
    hipMemcpy(&terminate_flag, d_terminate_flag, sizeof(int), hipMemcpyDeviceToHost);

    if (terminate_flag == 1)
    {
      break;

    }
    printf("Kernel time: %f seconds\n", time );
    hipMemcpy(G,d_G, n*n*sizeof(int), hipMemcpyDeviceToHost);


    //hipFree

  }

    hipFree(d_newG);
    hipFree(d_G);
    hipFree(d_w);


}


//kernel function
__global__ void calculateNewSpinKernel(int * M, int * newM, double * w, int n, int * flag)
{

  //indeces
  int row = blockIdx.x*blockDim.x+threadIdx.x;
  int col = blockIdx.y*blockDim.y+threadIdx.y;
  int thread_id = col* n + row;


  //guard for extra threads
  if(thread_id < n*n)
  {
  //add for loop back and implement grid stride
    for( int stride_thread_id = thread_id; stride_thread_id<n*n; stride_thread_id += (blockDim.x * gridDim.x))
    {

      double influence = 0;
      //coordinates
      int y = stride_thread_id / n;
      int x = stride_thread_id % n;

      for (int k=-2; k<=2;k++)
      {
        for(int l= -2; l<=2; l++)
        {
          influence += w[(2+k)*5+(2+l)] * M[((k + y + n) % n) * n + (l + x + n) % n];
        }
      }

      //influence float point error
      if(fabs(influence) < 10e-7)
      {
      newM[stride_thread_id] = M[stride_thread_id];
      }
      else if(influence>0)
      {
      if(M[stride_thread_id]!=1)
        *flag = 0;
        newM[stride_thread_id] = 1;
      }
      else if(influence<0)
      {
      if(M[stride_thread_id]!=-1)
        *flag = 0;
        newM[stride_thread_id] = -1;
      }
    }
  }
}

